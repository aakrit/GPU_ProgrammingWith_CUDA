// we use the __device__ prefix to mark funcitons as callable from threads running // on the device memory

// device function (using __device__ ) are called from the __global__ functions
/*
__device__ float device_fucntion(float x){
  return x + 5;
}
*/
//__device__ funcitons can call other device functions but not them selves

//OK
/*
__device__ float device_function_two(float y){
  return device_function(y)/ 2;
}
*/

//NOT  (currently not supported in CUDA)
/*
__device__ float device_function_two(float z){
  return z * device_fucntion_two(z-1);
}
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ int get_thread_index(void){
  return blockIdx.x * blockDim.x + threadIdx.x;
}
__device__ int get_fixedValue(void){
  return 10;
}
__global__ void gpuone(int *a){
  int index = get_thread_index();
  a[index] = get_fixedValue();
}
__global__ void gputwo(int *b){
  int index = get_fixedValue();
  b[index] = get_thread_index();
}
void printArray(int* host, int elements){
  printf("GPU array: \n");
  for(int i = 0; i < elements; i++){
    printf("%d ", host[i]);
  }
  printf("\n\n");
}

int main(void){
  int elements = 256;
  int bytes = elements * sizeof(int);

  int *host = 0, *device = 0;

  host = (int*) malloc(bytes);
  hipMalloc((void**)&device, bytes);

  int blockSize = 128;
  int gridSize = elements / blockSize;

  gpuone<<<gridSize, blockSize>>>(device);
  hipMemcpy(host, device, bytes, hipMemcpyDeviceToHost);

  printArray(host, elements);

  gputwo<<<gridSize, blockSize>>>(device);
  hipMemcpy(host, device, bytes, hipMemcpyDeviceToHost);

  printArray(host, elements);

  free(host);
  hipFree(device);
  return 0;

}


