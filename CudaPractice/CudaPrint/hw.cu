#include "hip/hip_runtime.h"
/*
 __global__ functions, or kernels, are like the main function from C. They are the first point of entry into a program which is executed by the GPU device. __global__ functions are qualified with the special CUDA keyword, __global__, which is prepended to their function signature.
 */

#include <stdio.h>
#include "util/cuPrintf.cu"

__global__ void gpu_greeting(void)
{
  cuPrintf("Hello, i'm your GPU talking\n");
}

int main(void){
  //run on CPU
  printf("Hi, this is your CPU computing\n");

  //initialize cuPrintf
  cudaPrintfInit();

  //launch kernel with single thread to greet from device
  gpu_greeting<<<3,3>>>(); //9 threads = 3 blocks with 3 threads eachs

  //display device greetings
  cudaPrintfDisplay();

  //clean up after cuPrintf
  cudaPrintfEnd();

  return 0;
}
