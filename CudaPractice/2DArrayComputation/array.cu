
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void kernel(int *array){

  int index_x = blockIdx.x * blockDim.x + threadIdx.x;
  int index_y = blockIdx.y * blockDim.y + threadIdx.y;

  //mapping 2D indcies to a 1D index in device memeory
  int grid_width = gridDim.x * blockDim.x;
  int index = index_y * grid_width + index_x;

  //map the two 2D black indices to a single linear, 1D block
  int result = blockIdx.y * gridDim.x + blockIdx.x;

  //write out the result
  array[index] = result;
}

int main(void){

  int num_elements_x = 16;
  int num_elements_y = 16;

  int num_bytes = num_elements_x * num_elements_y * sizeof(int);
  int *gpu_array = 0;
  int *cpu_array = 0;

  //allocate memeory for cpu and gpu
  cpu_array = (int*)malloc(num_bytes);
  hipMalloc((void**)&gpu_array, num_bytes);

  //create 2D 4x4 thread blocks
  dim3 block_size;
  block_size.x = 4;
  block_size.y = 4;

  //configure a 2D grid as well
  dim3 grid_size;
  grid_size.x = num_elements_x / block_size.x;
  grid_size.y = num_elements_y / block_size.y;

  //pass the grids to the kernel and have the gpu execute
  kernel<<<grid_size, block_size>>>(gpu_array);

  //copy results and inspect on the cpu
  hipMemcpy(cpu_array, gpu_array, num_bytes, hipMemcpyDeviceToHost);
  for(int row = 0; row < num_elements_y; ++row){
    for(int col = 0; col < num_elements_x; ++col){
      printf("%2d ", cpu_array[row * num_elements_x + col]);
    }
    printf("\n");
  }
  printf("\n");
  //free memeory
  free(cpu_array);
  hipFree(gpu_array);
}




