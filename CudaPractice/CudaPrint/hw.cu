#include "hip/hip_runtime.h"
/*
 __global__ functions, or kernels, are like the main function from C. They are the first point of entry into a program which is executed by the GPU device. __global__ functions are qualified with the special CUDA keyword, __global__, which is prepended to their function signature.
 */

#include <stdio.h>
#include "util/cuPrintf.cu"

static int threads = 3;
static int blocks = 3;

__global__ void gpu_greeting(void)
{
  cuPrintf("Hello, i'm your GPU talking\n");
}

int main(int argc, char** argv){
  //run on CPU
  printf("Hi, this is your CPU computing\n");
  if(argc > 1){
	threads = (int) atoi(argv[1]);
  }
  if(argc > 2){
	blocks = (int) atoi(argv[2]);
  }
  //initialize cuPrintf
  cudaPrintfInit();

  //launch kernel with single thread to greet from device
  gpu_greeting<<<blocks,threads>>>(); //9 threads = 3 blocks with 3 threads eachs

  //display device greetings
  cudaPrintfDisplay();

  //clean up after cuPrintf
  cudaPrintfEnd();

  return 0;
}
