

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

extern int j = 0;

__global__ void kernel(int *array, int j){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  array[index] = j;
  j++;
}

int main(void){
  int elements = 256;
  int bytes = elements * sizeof(int);

  //points to cpu and gpu arrays
  int *gpu_array = 0;
  int *cpu_array = 0;

  //malloc cpu array
  cpu_array = (int *)malloc(bytes);

  //cudaMalloc gpu array
  hipMalloc((void**) &gpu_array, bytes);

  int blockSize = 128;
  int gridSize = elements/ blockSize;

  kernel<<<gridSize, blockSize>>>(gpu_array, j);

  //copy to host
  hipMemcpy(cpu_array, gpu_array, bytes, hipMemcpyDeviceToHost);

  //print results
  for(int i = 0; i < elements; ++i){
    printf("%d ", cpu_array[i]);
  }
  printf("\n");
  //de_allocate memory
  free(cpu_array);
  hipFree(gpu_array);

}

