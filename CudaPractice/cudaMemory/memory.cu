
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

int main(void){

  int elements = 16;
  int bytes = elements * sizeof(int);

  int *device = 0;
  int *host = 0;

  host = (int*)malloc(bytes);

  hipMalloc((void**)&device, bytes);
  hipMemset(device, 0, bytes);
  hipMemcpy(host, device, bytes, hipMemcpyDeviceToHost);
  printf("\n");
  for(int i = 0; i < elements; ++i){
    printf("%d", host[i]);
  }
  /* can't access device without using host to copy into 1st
  printf("\n");
  for(int j = 0; j < elements; ++j){
    printf("%d", device[i]);
  }
  */
  free(host);

  hipFree(device);

  return 0;

}
